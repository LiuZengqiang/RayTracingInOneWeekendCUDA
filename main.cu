#include "hip/hip_runtime.h"
#include <float.h>

#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "camera.h"
#include "color.h"
#include "hittable_list.h"
#include "interval.h"
#include "material.h"
#include "ray.h"
#include "rtweekend.h"
#include "sphere.h"
#include "vec3.h"

__global__ void create_world(hittable_list** world, hiprandState* rand_state) {
  int id = getThreadId();
  if (id > 0) return;
  hiprand_init(1984, id, 0, rand_state);

  *world = new hittable_list();

  lambertian* ground_material = new lambertian(color(0.5, 0.5, 0.5));

  (*world)->add(new sphere(point3(0, -1000, 0), 1000, ground_material));

  for (int a = -11; a < 11; a++) {
    for (int b = -11; b < 11; b++) {
      float choose_mat = random_double(rand_state);

      point3 center(a + 0.9f * random_double(rand_state), 0.2f,
                    b + 0.9f * random_double(rand_state));
      // 根据随机数 choose_mat 的取值大小随机生成 diffuse, metal, glass 材质的球
      if ((center - point3(4, 0.2f, 0)).length() > 0.9f) {
        material* sphere_material;

        if (choose_mat < 0.8f) {
          // diffuse, 漫反射材质, 反射光线在交点法线方向存在散射(漫反射)和衰减
          auto albedo = random(rand_state) * random(rand_state);

          sphere_material = new lambertian(albedo);

          (*world)->add(new sphere(center, 0.2, sphere_material));
        } else if (choose_mat < 0.95) {
          // metal, 金属材质, 反射光线在理想反射光线方向存在散射(漫反射)和衰减
          auto albedo = random(0.5f, 1.0f, rand_state);
          auto fuzz = random_double(0.0f, 0.5f, rand_state);

          sphere_material = new metal(albedo, fuzz);
          (*world)->add(new sphere(center, 0.2f, sphere_material));
        } else {
          // glass, 玻璃材质, 反射光线存在折射,反射和衰减
          sphere_material = new dielectric(1.5);
          (*world)->add(new sphere(center, 0.2f, sphere_material));
        }
      }
    }
  }

  dielectric* material1 = new dielectric(1.5f);
  (*world)->add(new sphere(point3(0, 1, 0), 1.0f, material1));

  lambertian* material2 = new lambertian(color(0.4f, 0.2f, 0.1f));
  (*world)->add(new sphere(point3(-4, 1, 0), 1.0, material2));

  metal* material3 = new metal(color(0.7, 0.6, 0.5), 0.0f);
  (*world)->add(new sphere(point3(4, 1, 0), 1.0, material3));
}

__global__ void free_world(hittable_list** world) {
  (*world)->clear();
  delete *world;
}
int main() {
  hittable_list** d_world = nullptr;

  hiprandState* d_rand_state;
  checkCudaErrors(
      hipMallocManaged((void**)&d_rand_state, sizeof(hiprandState)));

  hipMalloc((void**)&d_world, 1 * sizeof(hittable_list*));
  create_world<<<1, 1>>>(d_world, d_rand_state);
  checkCudaErrors(hipDeviceSynchronize());

  /* 设置相机和输出图像的属性 */
  camera cam;
  cam.aspect_ratio = 16.0 / 9.0;  // 图像的长宽比
  cam.image_width = 800;          // 图像的宽(像素数)
  cam.samples_per_pixel = 100;    // 每个像素的采样光线数
  cam.max_depth = 50;             // 光线的最大深度

  cam.vfov = 20;                    // 视场角
  cam.lookfrom = point3(13, 2, 3);  // 相机位置
  cam.lookat = point3(0, 0, 0);     // 相机观察的点
  cam.vup = vec3(0, 1, 0);          // 相机上方向向量

  cam.defocus_angle = 0.6;  // 模拟实际相机的散射角度(以实现景深效果)
  cam.focus_dist = 10.0;  // 模拟实际相机的理想焦距(以实现景深效果)

  cam.render(d_world);
  checkCudaErrors(hipDeviceSynchronize());

  free_world<<<1, 1>>>(d_world);
  checkCudaErrors(hipDeviceSynchronize());

  hipFree(d_world);
  hipFree(d_rand_state);
  return 0;
}